#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"

PGM_IMG contrast_enhancement_g(PGM_IMG img_in)
{
    PGM_IMG result;
    int hist[256];
    
    result.w = img_in.w;
    result.h = img_in.h;
    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    
    histogram(hist, img_in.img, img_in.h * img_in.w, 256);
    histogram_equalization(result.img,img_in.img,hist,result.w*result.h, 256);
    return result;
}

PPM_IMG contrast_enhancement_c_rgb(PPM_IMG img_in)
{
    PPM_IMG result;
    int hist[256];
    
    result.w = img_in.w;
    result.h = img_in.h;
    result.img_r = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    result.img_g = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    result.img_b = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    
    histogram(hist, img_in.img_r, img_in.h * img_in.w, 256);
    histogram_equalization(result.img_r,img_in.img_r,hist,result.w*result.h, 256);
    histogram(hist, img_in.img_g, img_in.h * img_in.w, 256);
    histogram_equalization(result.img_g,img_in.img_g,hist,result.w*result.h, 256);
    histogram(hist, img_in.img_b, img_in.h * img_in.w, 256);
    histogram_equalization(result.img_b,img_in.img_b,hist,result.w*result.h, 256);

    return result;
}


PPM_IMG contrast_enhancement_c_yuv(PPM_IMG img_in)
{
    YUV_IMG yuv_med;
    PPM_IMG result;
    
    unsigned char * y_equ;
    int hist[256];
    
    yuv_med = rgb2yuv(img_in);
    y_equ = (unsigned char *)malloc(yuv_med.h*yuv_med.w*sizeof(unsigned char));
    
    histogram(hist, yuv_med.img_y, yuv_med.h * yuv_med.w, 256);
    histogram_equalization(y_equ,yuv_med.img_y,hist,yuv_med.h * yuv_med.w, 256);

    free(yuv_med.img_y);
    yuv_med.img_y = y_equ;
    
    result = yuv2rgb(yuv_med);
    free(yuv_med.img_y);
    free(yuv_med.img_u);
    free(yuv_med.img_v);
    
    return result;
}

PPM_IMG contrast_enhancement_c_hsl(PPM_IMG img_in)
{
    HSL_IMG hsl_med;
    PPM_IMG result;
    
    unsigned char * l_equ;
    int hist[256];

    hsl_med = rgb2hsl(img_in);
    l_equ = (unsigned char *)malloc(hsl_med.height*hsl_med.width*sizeof(unsigned char));

    histogram(hist, hsl_med.l, hsl_med.height * hsl_med.width, 256);
    histogram_equalization(l_equ, hsl_med.l,hist,hsl_med.width*hsl_med.height, 256);
    
    free(hsl_med.l);
    hsl_med.l = l_equ;

    result = hsl2rgb(hsl_med);
    free(hsl_med.h);
    free(hsl_med.s);
    free(hsl_med.l);
    return result;
}

__global__ void for_rgb2hsl(PPM_IMG img_in, HSL_IMG img_out)
{
    //__shared__ int temp[THREADS_PER_BLOCK];
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    float H, S, L;
    float var_r = ( (float)img_in.img_r[i]/255 );//Convert RGB to [0,1]
    float var_g = ( (float)img_in.img_g[i]/255 );
    float var_b = ( (float)img_in.img_b[i]/255 );
    float var_min = (var_r < var_g) ? var_r : var_g;
    var_min = (var_min < var_b) ? var_min : var_b;   //min. value of RGB
    float var_max = (var_r > var_g) ? var_r : var_g;
    var_max = (var_max > var_b) ? var_max : var_b;   //max. value of RGB
    float del_max = var_max - var_min;               //Delta RGB value
    
    L = ( var_max + var_min ) / 2;
    if ( del_max == 0 )//This is a gray, no chroma...
    {
        H = 0;         
        S = 0;    
    }
    else                                    //Chromatic data...
    {
        if ( L < 0.5 )
            S = del_max/(var_max+var_min);
        else
            S = del_max/(2-var_max-var_min );

        float del_r = (((var_max-var_r)/6)+(del_max/2))/del_max;
        float del_g = (((var_max-var_g)/6)+(del_max/2))/del_max;
        float del_b = (((var_max-var_b)/6)+(del_max/2))/del_max;
        if( var_r == var_max ){
            H = del_b - del_g;
        }
        else{       
            if( var_g == var_max ){
                H = (1.0/3.0) + del_r - del_b;
            }
            else{
                    H = (2.0/3.0) + del_g - del_r;
            }   
        }
        
    }
    
    if ( H < 0 )
        H += 1;
    if ( H > 1 )
        H -= 1;

    img_out.h[i] = H;
    img_out.s[i] = S;
    img_out.l[i] = (unsigned char)(L*255);

}

//Convert RGB to HSL, assume R,G,B in [0, 255]
//Output H, S in [0.0, 1.0] and L in [0, 255]
HSL_IMG rgb2hsl(PPM_IMG img_in)
{
    HSL_IMG img_out, d_img_out;// = (HSL_IMG *)malloc(sizeof(HSL_IMG));
    PPM_IMG d_img_in;

    // Allocate Device copies PPM_IMG img_in, YUV_IMG img_out
    hipMalloc((void**)&d_img_in,  sizeof(PPM_IMG));
    hipMalloc((void**)&d_img_out, sizeof(HSL_IMG)); 

    img_out.width  = img_in.w;
    img_out.height = img_in.h;
    img_out.h = (float *)malloc(img_in.w * img_in.h * sizeof(float));
    img_out.s = (float *)malloc(img_in.w * img_in.h * sizeof(float));
    img_out.l = (unsigned char *)malloc(img_in.w * img_in.h * sizeof(unsigned char));
    
    // Copy Inputs to Device
    hipMemcpy( &d_img_out, &img_out, sizeof(HSL_IMG), hipMemcpyHostToDevice );
    hipMemcpy( &d_img_in, &img_in, sizeof(PPM_IMG), hipMemcpyHostToDevice );
    
    // (11472 x 6429) = 73,753,488 approx 74 million pixels approx.
    for_rgb2hsl<<<img_in.w,img_in.h>>>(d_img_in,d_img_out);

    // Copy Device Result ---> Host copy of result
    hipMemcpy( &img_out, &d_img_out, sizeof(HSL_IMG), hipMemcpyDeviceToHost );

    //Sync b/w Host(CPU) and Device(GPU)    
    hipDeviceSynchronize();
    
    return img_out;
}

float Hue_2_RGB( float v1, float v2, float vH )             //Function Hue_2_RGB
{
    if ( vH < 0 ) vH += 1;
    if ( vH > 1 ) vH -= 1;
    if ( ( 6 * vH ) < 1 ) return ( v1 + ( v2 - v1 ) * 6 * vH );
    if ( ( 2 * vH ) < 1 ) return ( v2 );
    if ( ( 3 * vH ) < 2 ) return ( v1 + ( v2 - v1 ) * ( ( 2.0f/3.0f ) - vH ) * 6 );
    return ( v1 );
}

//Convert HSL to RGB, assume H, S in [0.0, 1.0] and L in [0, 255]
//Output R,G,B in [0, 255]

__global__ void for_hsl2rgb(HSL_IMG d_img_in, PPM_IMG d_result)
{
    float dHue_2_RGB;
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    float H = d_img_in.h[index];
    float S = d_img_in.s[index];
    float L = d_img_in.l[index]/255.0f;
    float var_1, var_2;
    
    unsigned char r,g,b;
    
    if ( S == 0 )
    {
        r = L * 255;
        g = L * 255;
        b = L * 255;
    }
    else
    {
        
        if ( L < 0.5 )
            var_2 = L * ( 1 + S );
        else
            var_2 = ( L + S ) - ( S * L );

        var_1 = 2 * L - var_2;


        if ( H < 0 ) H += 1;
        if ( H > 1 ) H -= 1;
        if ( ( 6 * H ) < 1 )
            dHue_2_RGB = ( var_1 + ( var_2 - var_1 ) * 6 * H );
        if ( ( 2 * H ) < 1 ) 
            dHue_2_RGB = ( var_2 );
        if ( ( 3 * H ) < 2 ) 
            dHue_2_RGB = ( var_1 + ( var_2 - var_1 ) * ( ( 2.0f/3.0f ) - H ) * 6 );
        dHue_2_RGB = ( var_1 );

        r = 255 * dHue_2_RGB;
        g = 255 * dHue_2_RGB;
        b = 255 * dHue_2_RGB;
    }
    d_result.img_r[index] = r;
    d_result.img_g[index] = g;
    d_result.img_b[index] = b;


}



PPM_IMG hsl2rgb(HSL_IMG img_in)
{

    PPM_IMG result, d_result;
    HSL_IMG d_img_in;

    // Allocate Device copies
    hipMalloc((void**)&d_img_in,  sizeof(HSL_IMG));
    hipMalloc((void**)&d_result, sizeof(PPM_IMG)); 
    
    result.w = img_in.width;
    result.h = img_in.height;
    result.img_r = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    result.img_g = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    result.img_b = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    
    // Copy Inputs to Device
    hipMemcpy( &d_img_in, &img_in, sizeof(HSL_IMG), hipMemcpyHostToDevice );
    hipMemcpy( &d_result, &result, sizeof(PPM_IMG), hipMemcpyHostToDevice );

    for_hsl2rgb<<<img_in.width,img_in.height>>>(d_img_in,d_result);

    // Copy Device Result ---> Host copy of result
    hipMemcpy( &result, &d_result, sizeof(PPM_IMG), hipMemcpyDeviceToHost );

    //Sync b/w Host(CPU) and Device(GPU)    
    hipDeviceSynchronize();

    return result;
}

//Declared on Device
__global__ void for_rgb2yuv(PPM_IMG d_img_in, YUV_IMG d_img_out)
{
    
    //__shared__ int temp[THREADS_PER_BLOCK];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    //temp[threadIdx.x] = a[index] * b[index];

    //__syncthreads();

    unsigned char r, g, b;
    unsigned char y, cb, cr;

   //for(i = 0; i < img_out.w*img_out.h; i ++)
    r = d_img_in.img_r[index];
    g = d_img_in.img_g[index];
    b = d_img_in.img_b[index];
    
    y  = (unsigned char)( 0.299*r + 0.587*g +  0.114*b);
    cb = (unsigned char)(-0.169*r - 0.331*g +  0.499*b + 128);
    cr = (unsigned char)( 0.499*r - 0.418*g - 0.0813*b + 128);
    
    d_img_out.img_y[index] = y;
    d_img_out.img_u[index] = cb;
    d_img_out.img_v[index] = cr;    
}

//Convert RGB to YUV, all components in [0, 255]
YUV_IMG rgb2yuv(PPM_IMG img_in)
{
    YUV_IMG img_out;
    //PPM_IMG *img_in;

    //int i;//, j;
    //unsigned char r, g, b;
    //unsigned char y, cb, cr;
    PPM_IMG d_img_in;
    YUV_IMG d_img_out;

    // Allocate Device copies PPM_IMG img_in, YUV_IMG img_out
    hipMalloc((void**)&d_img_in,  sizeof(PPM_IMG));
    hipMalloc((void**)&d_img_out, sizeof(YUV_IMG));    
    
    img_out.w = img_in.w;
    img_out.h = img_in.h;
    img_out.img_y = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);
    img_out.img_u = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);
    img_out.img_v = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);

    // Copy Inputs to Device
    hipMemcpy( &d_img_out, &img_out, sizeof(YUV_IMG), hipMemcpyHostToDevice );
    hipMemcpy( &d_img_in, &img_in, sizeof(PPM_IMG), hipMemcpyHostToDevice );

    // (11472 x 6429) = 73,753,488 approx 74 million pixels approx.
    for_rgb2yuv<<<img_out.w,img_out.h>>>(d_img_in,d_img_out);

    // Copy Device Result ---> Host copy of result
    hipMemcpy( &img_out, &d_img_out, sizeof(YUV_IMG), hipMemcpyDeviceToHost );

    //Sync b/w Host(CPU) and Device(GPU)    
    hipDeviceSynchronize();
    return img_out;
}

//Declared on Device
__global__ void for_yuv2rgb(YUV_IMG d_img_in, PPM_IMG d_img_out)
{
    //__shared__ int temp[THREADS_PER_BLOCK];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int rt,gt,bt;
    int y, cb, cr;

    y  = (int)d_img_in.img_y[index];
    cb = (int)d_img_in.img_u[index] - 128;
    cr = (int)d_img_in.img_v[index] - 128;
    
    rt  = (int)( y + 1.402*cr);
    gt  = (int)( y - 0.344*cb - 0.714*cr);
    bt  = (int)( y + 1.772*cb);


    if(rt > 255)
        rt = 255;
    if(rt < 0)
        rt = 0;

    d_img_out.img_r[index] = rt;

    if(gt > 255)
        gt = 255;
    if(gt < 0)
        gt = 0;

    d_img_out.img_g[index] = gt;
    
    if(bt > 255)
        bt = 255;
    if(bt < 0)
        bt = 0;

    d_img_out.img_b[index] = bt;
    
}

//Convert YUV to RGB, all components in [0, 255]
PPM_IMG yuv2rgb(YUV_IMG img_in)
{
    PPM_IMG img_out, d_img_out;
    YUV_IMG d_img_in;

    // Allocate Device copies PPM_IMG img_in, YUV_IMG img_out
    hipMalloc((void**)&d_img_in,  sizeof(YUV_IMG));
    hipMalloc((void**)&d_img_out, sizeof(PPM_IMG));    
    

    img_out.w = img_in.w;
    img_out.h = img_in.h;
    img_out.img_r = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);
    img_out.img_g = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);
    img_out.img_b = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);

    // Copy Inputs to Device
    hipMemcpy( &d_img_out, &img_out, sizeof(PPM_IMG), hipMemcpyHostToDevice );
    hipMemcpy( &d_img_in, &img_in, sizeof(YUV_IMG), hipMemcpyHostToDevice );

    // (11472 x 6429) = 73,753,488 approx 74 million pixels approx.
    for_yuv2rgb<<<img_out.w,img_out.h>>>(d_img_in,d_img_out);
    
    // Copy Device Result ---> Host copy of result
    hipMemcpy( &img_out, &d_img_out, sizeof(PPM_IMG), hipMemcpyDeviceToHost );

    //Sync b/w Host(CPU) and Device(GPU)    
    hipDeviceSynchronize();

    return img_out;
}
